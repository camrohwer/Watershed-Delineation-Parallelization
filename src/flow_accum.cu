#include "hip/hip_runtime.h"
#include <iostream>
#include <gdal_priv.h>
#include <hip/hip_runtime.h>
#include <cmath>
#include <cstdio>
#include <cstdlib>

#define THREADCELLS 4
#define BLOCK_SIZE 8

__constant__ int offsetX[9] = {0, -1,  0,  1,  1,  1,  0, -1, -1};
__constant__ int offsetY[9] = {0, -1, -1, -1,  0,  1,  1,  1,  0};
                            // 0,  1,  2,  3,  4,  5,  6,  7,  8 


//two nested for loops - handle in the neighborhood sequentially
//fourth array - takes place of the cells - used for writing final to? cumulative?
__global__ void flowAccumKernel(int* gpuAccum, int* gpuOldFlow, int* gpuNewFlow, int * flowDir, int* gpuStop, int N, int M){
    int i = THREADCELLS * (blockIdx.y * blockDim.y + threadIdx.y);
    int j = THREADCELLS * (blockIdx.x * blockDim.x + threadIdx.x);

    for (int r = i; r < i + THREADCELLS && r < N; r++){
        for (int s = j; s < j + THREADCELLS && s < M; s++){
            int curFlow = gpuOldFlow[r * M + s];
            if (curFlow > 0){
                gpuOldFlow[r * M + s] = 0;
                int cellFlowDir = flowDir[r * M + s]; 
                if (cellFlowDir == 0) break;
                int newR = r + offsetY[cellFlowDir];
                int newS = s + offsetX[cellFlowDir];

                if (newR >= 0 && newR < N && newS >= 0 && newS < M) {
                    atomicAdd(&gpuNewFlow[newR * M + newS], curFlow);
                    atomicAdd(&gpuAccum[newR * M + newS], curFlow);
                    atomicOr(gpuStop, 1);
                }
            } 
        }
    }
}

int main(int argc, char* argv[]){
    if (argc < 3){
        std::cout << "Please provide a filepath for input and output raster" << std::endl;
        return -1;
    }
    // register drivers to open raster data
    GDALAllRegister();
    
    // Open DEM dataset
    const char* input = argv[1];
    GDALDataset* D8Dataset  = (GDALDataset*) GDALOpen(input, GA_ReadOnly);

    if (D8Dataset == NULL) {
        std::cerr << "Error opening flow direction file." << std::endl;
        return -1;
    }

    //Get projection info
    const char* projection = D8Dataset->GetProjectionRef();
    if (projection == nullptr){
        std::cerr<< "Error: Could not retrieve projection information" << std::endl;
        GDALClose(D8Dataset);
        return -1;
    }
    double geoTransform[6];

    if (D8Dataset->GetGeoTransform(geoTransform) != CE_None){
        std::cerr << "Error reading geo-transform" << std::endl;
        GDALClose(D8Dataset);
        return -1;
    }

    //create output raster for flow accumulation
    const char *outputFilename = argv[2];
    //Geotiff Driver
    GDALDriver *poDriver = GetGDALDriverManager()->GetDriverByName("GTiff");
    //32int Empty raster with same dims as input
    GDALDataset *flowAccumDataset = poDriver->Create(outputFilename,
                                                    D8Dataset->GetRasterXSize(),
                                                    D8Dataset->GetRasterYSize(),
                                                    1, GDT_Int32, NULL);

    //Set projection of output
    flowAccumDataset->SetProjection(projection);
    flowAccumDataset->SetGeoTransform(geoTransform);

    //Raster size to use with Malloc and device mem
    int width = D8Dataset->GetRasterXSize();
    int height = D8Dataset->GetRasterYSize();
    int *flowDir = (int *)CPLMalloc(sizeof(int) * width * height);

    //populate demData dynamically allocated memory
    CPLErr err = D8Dataset->GetRasterBand(1)->RasterIO(GF_Read, 0, 0, width, height, flowDir, width, height, GDT_Int32, 0, 0);
    if (err != CE_None){
        std::cerr << "Error reading DEM data: " << CPLGetLastErrorMsg() << std::endl;
        return -1;
    }

    //allocating device mem
    int *d_oldFlow, *d_newFlow, *d_flowDir, *d_accum, *d_stopFlag;
    
    // Allocate memory for d_oldFlow on device
    if (hipMalloc(&d_oldFlow, width * height * sizeof(int)) != hipSuccess) {
        std::cerr << "Error allocating memory for Old Flow on device" << std::endl;
        return -1;
    }
    // Allocate memory for d_newFlow on device
    if (hipMalloc(&d_newFlow, width * height * sizeof(int)) != hipSuccess) {
        std::cerr << "Error allocating memory for New Flow on device" << std::endl;
        return -1;
    }
    // Allocate memory for d_flowDir on device
    if (hipMalloc(&d_flowDir, width * height * sizeof(int)) != hipSuccess) {
        std::cerr << "Error allocating memory for Flow Direction on device" << std::endl;
        return -1;
    }
      // Allocate memory for d_flowDir on device
    if (hipMalloc(&d_accum, width * height * sizeof(int)) != hipSuccess) {
        std::cerr << "Error allocating memory for Flow Direction on device" << std::endl;
        return -1;
    }
    // Allocate memory for d_stopFlag on device
    if (hipMalloc(&d_stopFlag, sizeof(int)) != hipSuccess) {
        std::cerr << "Error allocating memory for Stop Flag on device" << std::endl;
        return -1;
    }
        
    //copy flow direction data to device
    hipError_t memcpy_err_flowDir = hipMemcpy(d_flowDir, flowDir, sizeof(int) * width * height, hipMemcpyHostToDevice);
    if (memcpy_err_flowDir != hipSuccess){
        std::cerr << "Error copying data to device: " << hipGetErrorString(memcpy_err_flowDir) << std::endl;
        return -1;
    }
    int* hostOldFlow = new int[width * height];
    int* hostNewFlow = new int [width*height];
    for (int i = 0; i < width * height; ++i) hostOldFlow[i] = 1, hostNewFlow[i] = 0;

    hipMemcpy(d_oldFlow, hostOldFlow, width * height * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_newFlow, hostNewFlow, width * height * sizeof(int), hipMemcpyHostToDevice);
    hipMemset(d_accum, 0, sizeof(int) * width * height);   

    //define grid and block size
    dim3 blockSize(BLOCK_SIZE, BLOCK_SIZE);
    dim3 gridSize((width + blockSize.x - 1) / blockSize.x, (height + blockSize.y - 1) / blockSize.y);

    int x = 0;
    int *stopFlag = new int(0);

    do{
        printf("Iteration: %d\n", x++);
        *stopFlag = 0;
        hipMemcpy(d_stopFlag, stopFlag, sizeof(int), hipMemcpyHostToDevice);

        flowAccumKernel<<<gridSize, blockSize>>>(d_accum, d_oldFlow, d_newFlow, d_flowDir, d_stopFlag, height, width);

        hipError_t kernelErr = hipGetLastError();
        if (kernelErr != hipSuccess){
            std::cerr << "Error launching kernel: " << hipGetErrorString(kernelErr) << std::endl;
            return -1;
        }
        hipDeviceSynchronize();
        hipMemcpy(stopFlag, d_stopFlag, sizeof(int), hipMemcpyDeviceToHost);

        int *temp = d_oldFlow;
        d_oldFlow = d_newFlow;
        d_newFlow = temp;
        hipMemset(d_newFlow, 0, sizeof(int) * width * height);

        //early termination to prevent infite looping if error in flow direction data
        if (x == 500){
            break;
        }
    } while (*stopFlag != 0);

    int *hostflowAccumulationData = (int *)CPLMalloc(sizeof(int) * width * height);
    hipMemcpy(hostflowAccumulationData, d_accum, sizeof(int) * width * height, hipMemcpyDeviceToHost);

    err = flowAccumDataset->GetRasterBand(1)->RasterIO(GF_Write, 0, 0, width, height,
        hostflowAccumulationData, width, height, GDT_Int32, 0, 0);
    if (err != CE_None) {
        std::cerr << "Error writing flow accumulation data: " << CPLGetLastErrorMsg() << std::endl;
        return -1;
    }
    
    hipFree(d_oldFlow); hipFree(d_newFlow); hipFree(d_flowDir);
    hipFree(d_stopFlag); hipFree(d_accum);
    CPLFree(flowDir);
    GDALClose(D8Dataset); GDALClose(flowAccumDataset);
    delete[] hostOldFlow; delete[] hostNewFlow;
    return 0;
}