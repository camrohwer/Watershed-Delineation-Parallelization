#include "hip/hip_runtime.h"
#include <iostream>
#include <gdal_priv.h>
#include <hip/hip_runtime.h>
#include <cmath>
#include <cstdio>
#include <cstdlib>

#define THREADCELLS 4
#define BLOCK_SIZE 8
#define TILE_SIZE 8

__constant__ int offsetX[9] = {0, -1,  0,  1,  1,  1,  0, -1, -1};
__constant__ int offsetY[9] = {0, -1, -1, -1,  0,  1,  1,  1,  0};
                            // 0,  1,  2,  3,  4,  5,  6,  7,  8 

__device__ int getTiledIndex(int row, int col, int rows, int cols, int tile_size){
    if (row< 0 || row >= rows || col < 0 || col >= cols) return -1;

    int tiles_per_row = (cols + tile_size - 1) / tile_size;
    int tile_x = col / tile_size;
    int tile_y = row / tile_size;

    int offset_x = col % tile_size;
    int offset_y = row % tile_size;

    int tile_index = (tile_y * tiles_per_row + tile_x) * tile_size * tile_size;
    int local_index = offset_y * tile_size + offset_x;
    
    return tile_index + local_index;
}

__global__ void rowToTiled( int* input, int* output, int rows, int cols, int tile_size) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
     
    if (x < cols && y < rows) {
        int tiled_index = getTiledIndex(y, x, rows, cols, tile_size);
        output[tiled_index] = input[y * cols + x];
    }
}

__global__ void tiledToRow(int* input, int* output, int rows, int cols, int tile_size) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < cols && y < rows) {
        int tiled_index = getTiledIndex(y, x, rows, cols, tile_size);
        output[y * cols + x] = input[tiled_index];
    }
}

//Each thread handles a THREADCELLS x THREADCELLS neighbourhood
__global__ void flowAccumKernel(int* gpuAccum, int* gpuOldFlow, int* gpuNewFlow, const int * flowDir, int* gpuStop, const int N, const int M){
    int i = THREADCELLS * (blockIdx.y * blockDim.y + threadIdx.y);
    int j = THREADCELLS * (blockIdx.x * blockDim.x + threadIdx.x);
    
    for (int r = i; r < i + THREADCELLS && r < N; r++){
        for (int s = j; s < j + THREADCELLS && s < M; s++){
            int curFlow = gpuOldFlow[getTiledIndex(r, s, N, M, TILE_SIZE)];
            if (curFlow > 0){
                gpuOldFlow[getTiledIndex(r, s, N, M, TILE_SIZE)] = 0;
                int cellFlowDir = flowDir[getTiledIndex(r, s, N, M, TILE_SIZE)]; 
                if (cellFlowDir == 0) continue;
                int newR = r + offsetY[cellFlowDir];
                int newS = s + offsetX[cellFlowDir];

                int valid = (newR >= 0 && newR < N && newS >= 0 && newS < M);
                int new_idx = getTiledIndex(newR, newS, N, M, TILE_SIZE);

                if (valid && new_idx != -1){
                    atomicAdd(&gpuNewFlow[new_idx], valid * curFlow);
                    atomicAdd(&gpuAccum[new_idx], valid * curFlow);
                    atomicOr(gpuStop, 1);
                }
            } 
        }
    }
}

int main(int argc, char* argv[]){
    // FlowDir, FlowAccum
    if (argc < 3){
        std::cout << "Please provide a filepath for input and output raster" << std::endl;
        return -1;
    }
    // register drivers to open raster data
    GDALAllRegister();
    
    // Open DEM dataset
    const char* input = argv[1];
    GDALDataset* D8Dataset  = (GDALDataset*) GDALOpen(input, GA_ReadOnly);

    if (D8Dataset == NULL) {
        std::cerr << "Error opening flow direction file." << std::endl;
        return -1;
    }

    //Get projection info
    const char* projection = D8Dataset->GetProjectionRef();
    if (projection == nullptr){
        std::cerr<< "Error: Could not retrieve projection information" << std::endl;
        GDALClose(D8Dataset);
        return -1;
    }

    double geoTransform[6];
    if (D8Dataset->GetGeoTransform(geoTransform) != CE_None){
        std::cerr << "Error reading geo-transform" << std::endl;
        GDALClose(D8Dataset);
        return -1;
    }

    //create output raster for flow accumulation
    const char *outputFilename = argv[2];
    //Geotiff Driver
    GDALDriver *poDriver = GetGDALDriverManager()->GetDriverByName("GTiff");
    //32int Empty raster with same dims as input
    GDALDataset *flowAccumDataset = poDriver->Create(outputFilename,
                                                    D8Dataset->GetRasterXSize(),
                                                    D8Dataset->GetRasterYSize(),
                                                    1, GDT_Int32, NULL);

    //Set projection of output
    flowAccumDataset->SetProjection(projection);
    flowAccumDataset->SetGeoTransform(geoTransform);

    //Raster size to use with Malloc and device mem
    int width = D8Dataset->GetRasterXSize();
    int height = D8Dataset->GetRasterYSize();
    int *flowDir = (int *)CPLMalloc(sizeof(int) * width * height);

    //populate demData dynamically allocated memory
    CPLErr err = D8Dataset->GetRasterBand(1)->RasterIO(GF_Read, 0, 0, width, height, flowDir, width, height, GDT_Int32, 0, 0);
    if (err != CE_None){
        std::cerr << "Error reading DEM data: " << CPLGetLastErrorMsg() << std::endl;
        return -1;
    }

    //allocating device mem
    int *d_oldFlow, *d_newFlow, *d_flowDir, *d_flowDirTiled, *d_accum, *d_stopFlag;
    
    // Allocate memory for d_oldFlow on device
    if (hipMalloc(&d_oldFlow, width * height * sizeof(int)) != hipSuccess) {
        std::cerr << "Error allocating memory for Old Flow on device" << std::endl;
        return -1;
    }
    // Allocate memory for d_newFlow on device
    if (hipMalloc(&d_newFlow, width * height * sizeof(int)) != hipSuccess) {
        std::cerr << "Error allocating memory for New Flow on device" << std::endl;
        return -1;
    }
    // Allocate memory for d_flowDir on device
    if (hipMalloc(&d_flowDir, width * height * sizeof(int)) != hipSuccess) {
        std::cerr << "Error allocating memory for Flow Direction on device" << std::endl;
        return -1;
    }
    if (hipMalloc(&d_flowDirTiled, width * height * sizeof(int)) != hipSuccess) {
        std::cerr << "Error allocating memory for Flow Direction Tiled on device" << std::endl;
        return -1;
    }
      // Allocate memory for d_accum on device
    if (hipMalloc(&d_accum, width * height * sizeof(int)) != hipSuccess) {
        std::cerr << "Error allocating memory for Flow Direction on device" << std::endl;
        return -1;
    }
    // Allocate memory for d_stopFlag on device
    if (hipMalloc(&d_stopFlag, sizeof(int)) != hipSuccess) {
        std::cerr << "Error allocating memory for Stop Flag on device" << std::endl;
        return -1;
    }
        
    //copy flow direction data to device
    hipError_t memcpy_err_flowDir = hipMemcpy(d_flowDir, flowDir, sizeof(int) * width * height, hipMemcpyHostToDevice);
    if (memcpy_err_flowDir != hipSuccess){
        std::cerr << "Error copying data to device: " << hipGetErrorString(memcpy_err_flowDir) << std::endl;
        return -1;
    }

    int* hostOldFlow = new int[width * height];
    int* hostNewFlow = new int [width*height];
    for (int i = 0; i < width * height; ++i) hostOldFlow[i] = 1, hostNewFlow[i] = 0;

    hipMemcpy(d_oldFlow, hostOldFlow, width * height * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_newFlow, hostNewFlow, width * height * sizeof(int), hipMemcpyHostToDevice);
    hipMemset(d_accum, 0, sizeof(int) * width * height);   

    //define grid and block size
    dim3 blockSize(BLOCK_SIZE, BLOCK_SIZE);
    dim3 gridSize((width + blockSize.x - 1) / blockSize.x, (height + blockSize.y - 1) / blockSize.y);

    rowToTiled<<<gridSize, blockSize>>>(d_flowDir, d_flowDirTiled, height, width, TILE_SIZE);
    hipDeviceSynchronize();

    hipError_t kernel_err = hipGetLastError();
    if (kernel_err != hipSuccess){
        std::cerr << "Cuda kernel launch error: " << hipGetErrorString(kernel_err) << std::endl;
        return -1;
    }

    int iters = 0;
    int *stopFlag = new int(0);

    do{
        printf("Kernel iteration: %d\n", iters++ + 1);
        *stopFlag = 0;
        hipMemcpy(d_stopFlag, stopFlag, sizeof(int), hipMemcpyHostToDevice);

        flowAccumKernel<<<gridSize, blockSize>>>(d_accum, d_oldFlow, d_newFlow, d_flowDirTiled, d_stopFlag, height, width);

        hipError_t kernelErr = hipGetLastError();
        if (kernelErr != hipSuccess){
            std::cerr << "Error launching kernel: " << hipGetErrorString(kernelErr) << std::endl;
            return -1;
        }
        hipDeviceSynchronize();
        hipMemcpy(stopFlag, d_stopFlag, sizeof(int), hipMemcpyDeviceToHost);

        int *temp = d_oldFlow;
        d_oldFlow = d_newFlow;
        d_newFlow = temp;
        hipMemset(d_newFlow, 0, sizeof(int) * width * height);
    } while (*stopFlag != 0 && iters < 15000);

    int *hostflowAccumulationData = (int *)CPLMalloc(sizeof(int) * width * height);
    tiledToRow<<<gridSize, blockSize>>>(d_accum, d_oldFlow, height, width, TILE_SIZE);
    hipMemcpy(hostflowAccumulationData, d_oldFlow, sizeof(int) * width * height, hipMemcpyDeviceToHost); //temp use of oldFlow to hold converted row order format matrix before writing

    err = flowAccumDataset->GetRasterBand(1)->RasterIO(GF_Write, 0, 0, width, height,
        hostflowAccumulationData, width, height, GDT_Int32, 0, 0);
    if (err != CE_None) {
        std::cerr << "Error writing flow accumulation data: " << CPLGetLastErrorMsg() << std::endl;
        return -1;
    }
    
    //perform cleanup
    hipFree(d_oldFlow); hipFree(d_newFlow); hipFree(d_flowDir);
    hipFree(d_stopFlag); hipFree(d_accum);
    CPLFree(flowDir);
    GDALClose(D8Dataset); GDALClose(flowAccumDataset);
    delete[] hostOldFlow; delete[] hostNewFlow;
    return 0;
}