#include <cstdio>
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void hello_from_gpu(){
    printf("Hello from kernel!\n");
}

int main() {
    hello_from_gpu<<<1, 1>>>();

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("Kernel launch failed: %s\n", hipGetErrorString(err));
        return -1;
    }

    //wait for kernel to finish
    hipDeviceSynchronize();
    std::cout << "Hello from host!" <<std::endl;
    return 0;
}
