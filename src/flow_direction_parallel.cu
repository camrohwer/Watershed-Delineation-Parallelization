#include "hip/hip_runtime.h"
#include <iostream>
#include <gdal_priv.h>
#include <hip/hip_runtime.h>
#include <cmath>
#include <cstdio>
#include <cstdlib>

#define BLOCK_DIM_X 8
#define BLOCK_DIM_Y 8
#define FLOW_NODATA -1

__constant__ int offsetX[8] = { -1, 0, 1, 0, -1, 1, 1, -1 };
__constant__ int offsetY[8] = { 0, -1, 0, 1, -1, -1, 1, 1 };
__constant__ int direction[8] = { 8, 2, 4, 6, 1, 3, 5, 7 }; 

__global__ void flowDirectionKernel(int* dem, int* flow_dir, int width, int height) {
    //unique thread index
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x <= 0 || x >= width - 1 || y <= 0 || y >= height - 1) return; //skip boundary 
    
    __shared__ int sharedDem[BLOCK_DIM_Y + 2][BLOCK_DIM_X + 2];

    int tx = threadIdx.x + 1;
    int ty = threadIdx.y + 1;

    sharedDem[ty][tx] = dem[y * width + x];

    //left padding 
    if (threadIdx.x == 0 && x > 0){
        sharedDem[ty][0] = dem[y * width + (x - 1)];
    }
    //right padding
    if (threadIdx.x == blockDim.x - 1 && x < width - 1) {
        sharedDem[ty][tx + 1] = dem[y * width + (x + 1)];
    }
    //top padding
    if (threadIdx.y == 0 && y > 0){ 
        sharedDem[0][tx] = dem[(y - 1) * width + x];
    }
    //bottom padding
    if (threadIdx.y == blockDim.y - 1 && y < height - 1){
        sharedDem[ty + 1][tx] = dem[(y + 1) * width + x];
    }
    //top left corner
    if (threadIdx.x == 0 && threadIdx.y == 0 && x > 0 && y > 0) {
    sharedDem[0][0] = dem[(y - 1) * width + (x - 1)];
    }
    //top right corner
    if (threadIdx.x == blockDim.x - 1 && threadIdx.y == 0 && x < width - 1 && y > 0) {
        sharedDem[0][tx + 1] = dem[(y - 1) * width + (x + 1)];
    }
    //bottom left corner
    if (threadIdx.x == 0 && threadIdx.y == blockDim.y - 1 && x > 0 && y < height - 1) {
        sharedDem[ty + 1][0] = dem[(y + 1) * width + (x - 1)];
    }
    // bottom right corner
    if (threadIdx.x == blockDim.x - 1 && threadIdx.y == blockDim.y - 1 && x < width - 1 && y < height - 1) {
        sharedDem[ty + 1][tx + 1] = dem[(y + 1) * width + (x + 1)];
    }

    __syncthreads();

    int centre = sharedDem[ty][tx]; //get dem value at current pixel
    int lowest = centre;
    int dir = FLOW_NODATA;

    for (int i = 0; i < 8; i++) {
        int n = sharedDem[ty + offsetY[i]][tx + offsetX[i]];

        if (n < lowest){
            lowest = n;
            dir = direction[i];
        }
    }
    if (dir != FLOW_NODATA){
        flow_dir[y * width + x] = dir;
    }
}

int main(int argc, char* argv[]) {
    //checks for input file passed as arg
    if (argc < 3){
        std::cout << "Please provide a filepath for input and output raster" << std::endl;
        return -1;
    }
    // register drivers to open raster data
    GDALAllRegister();
    
    // Open DEM dataset
    const char* input = argv[1];
    GDALDataset* demDataset  = (GDALDataset*) GDALOpen(input, GA_ReadOnly);

    if (demDataset == nullptr) {
        std::cerr << "Error opening DEM file." << std::endl;
        return -1;
    }

    const char* projection = demDataset->GetProjectionRef();
    double geoTransform[6];

    if (demDataset->GetGeoTransform(geoTransform) != CE_None){
        std::cerr << "Error reading geo-transfor" << std::endl;
        GDALClose(demDataset);
        return -1;
    }

    //create output raster for flow direction
    const char *outputFilename = argv[2];
    //Geotiff Driver
    GDALDriver *poDriver = GetGDALDriverManager()->GetDriverByName("GTiff");
    //32int Empty raster with same dims as input
    GDALDataset *flowDirDataset = poDriver->Create(outputFilename,
                                                    demDataset->GetRasterXSize(),
                                                    demDataset->GetRasterYSize(),
                                                    1, GDT_Int32, NULL);

    flowDirDataset->SetProjection(projection);
    flowDirDataset->SetGeoTransform(geoTransform);

    //Raster size to use with Malloc and device mem
    int width = demDataset->GetRasterXSize();
    int height = demDataset->GetRasterYSize();
    float *demData = (float *)CPLMalloc(sizeof(float) * width * height);

    //populate demData dynamically allocated memory
    CPLErr err = demDataset->GetRasterBand(1)->RasterIO(GF_Read, 0, 0, width, height, demData, width, height, GDT_Float32, 0, 0);
    if (err != CE_None){
        std::cerr << "Error reading DEM data: " << CPLGetLastErrorMsg() << std::endl;
        return -1;
    }

    int *flowDirData = (int *)CPLMalloc(sizeof(int) * width * height);

    hipStream_t stream;
    hipStreamCreate(&stream);

    //allocating device mem
    int *d_demData;
    int *d_flowDirData;

    if (hipMalloc(&d_demData, sizeof(int) * width * height) != hipSuccess) {
        std::cerr << "Error allocating memory for DEM on device." << std::endl;
        return -1;
    }

    if (hipMalloc(&d_flowDirData, sizeof(int) * width * height) != hipSuccess) {
        std::cerr << "Error allocating memory for flow direction on device." << std::endl;
        hipFree(d_demData); // Free already allocated memory
        return -1;
    }

    //copy DEM data to device
    hipError_t memcpy_err = hipMemcpyAsync(d_demData, demData, sizeof(int) * width * height, hipMemcpyHostToDevice);
    if (memcpy_err != hipSuccess){
        std::cerr << "Error copying data to device: " << hipGetErrorString(memcpy_err) << std::endl;
        return -1;
    }

    //define grid and block size
    dim3 blockSize(BLOCK_DIM_X, BLOCK_DIM_Y);
    dim3 gridSize((width + blockSize.x - 1) / blockSize.x, (height + blockSize.y - 1) / blockSize.y);

    // Launch the CUDA kernel
    flowDirectionKernel<<<gridSize, blockSize, 0, stream>>>(d_demData, d_flowDirData, width, height);
    hipDeviceSynchronize();

    hipError_t kernel_err = hipGetLastError();
    if (kernel_err != hipSuccess){
        std::cerr << "Cuda kernel launch error: " << hipGetErrorString(kernel_err) << std::endl;
        return -1;
    }

    // Copy flow direction data back to host
    hipMemcpyAsync(flowDirData, d_flowDirData, sizeof(int) * width * height, hipMemcpyDeviceToHost);
    hipStreamSynchronize(stream);
    
    // Write flow direction data to the output dataset
    err = flowDirDataset->GetRasterBand(1)->RasterIO(GF_Write, 0, 0, width, height, flowDirData, width, height, GDT_Int32, 0, 0);
    if (err != CE_None) {
        std::cerr << "Error writing flow direction data: " << CPLGetLastErrorMsg() << std::endl;
        return -1;
    }

    // Cleanup
    CPLFree(demData);
    CPLFree(flowDirData);
    hipFree(d_demData);
    hipFree(d_flowDirData);
    GDALClose(demDataset);
    GDALClose(flowDirDataset);

    std::cout << "Flow direction calculated and saved to " << outputFilename << std::endl;
    return 0;
}
