#include <cstdio>
#include <hip/hip_runtime.h>

__global__ void hello_from_gpu(){
    printf("Hellow World!\n");
}

int main() {
    hello_from_gpu<<<1, 10>>>();

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("Kernel launch failed: %s\n", hipGetErrorString(err));
        return -1;
    }

    //wait for kernel to finish
    hipDeviceSynchronize();
    return 0;
}
