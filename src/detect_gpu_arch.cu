#include <iostream>
#include <hip/hip_runtime.h>

int main() {
    hipDeviceProp_t prop;
    int device;
    hipGetDevice(&device);
    hipGetDeviceProperties(&prop, device);
    std::cout << prop.major << prop.minor << std::endl;
    return 0;
}
