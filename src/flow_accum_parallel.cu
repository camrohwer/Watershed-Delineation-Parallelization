#include "hip/hip_runtime.h"
#include <iostream>
#include <gdal_priv.h>
#include <hip/hip_runtime.h>
#include <cmath>
#include <cstdio>
#include <cstdlib>

#define BLOCK_SIZE 16

__global__ void flowAccumKernel(int* gpuOldFlow, int* gpuNewFlow, int * flowDir, bool* gpuStop, int N, int M){
    //__shared__ int sharedOldFlow[BLOCK_SIZE][BLOCK_SIZE];
    //*gpuStop = false;
    int x = blockIdx.x * blockDim.x + threadIdx.x; 
    int y = blockIdx.y * blockDim.x + threadIdx.y;
    
    if (x >= M || y >= N){
        return;
    }
    int idx = y * M + x;
    //load flow data into shared memory
    //sharedOldFlow[x][y] = gpuOldFlow[idx];
    //__syncthreads();

    int flow = gpuOldFlow[y* M + x];
    //printf("(%d, %d) - %d  ",x,y, flow);
    if (flow <= 0 || flowDir[idx] == 0) return;
    if (flow > 0){
        gpuOldFlow[y* M + x] = 0;
        int dir = flowDir[idx];

        int targetX = x;
        int targetY = y;
        switch (dir){
            case 1: //northwest
                targetY -=1;
                targetX -=1;
                break;
            case 2: //north
                targetY -=1;
                break;
            case 3: //northeast
                targetY -=1;
                targetX +=1;
                break;
            case 4: //east
                targetX += 1;
                break;
            case 5: //southeast
                targetY +=1;
                targetX +=1;
                break;
            case 6: //south
                targetY +=1;
                break;
            case 7: //southwest
                targetY +=1;
                targetX -=1;
                break;
            case 8: //west
                targetX -=1;
                break;
        }
        //printf("--%d, %d--", targetX, targetY);

        if (targetX >= 0 && targetX < M && targetY >= 0 && targetY < N){
            atomicAdd(&gpuNewFlow[targetY * M + targetX], flow);
            *gpuStop = true;
            //printf("adjustment");
        }
    }



}

int main(int argc, char* argv[]){

    //checks for input file passed as arg
    if (argc < 3){
        std::cout << "Please provide a filepath for input and output raster" << std::endl;
        return -1;
    }
    // register drivers to open raster data
    GDALAllRegister();
    
    // Open DEM dataset
    const char* input = argv[1];
    GDALDataset* D8Dataset  = (GDALDataset*) GDALOpen(input, GA_ReadOnly);

    if (D8Dataset == NULL) {
        std::cerr << "Error opening flow direction file." << std::endl;
        return -1;
    }

    //create output raster for flow accumulation
    const char *outputFilename = argv[2];
    //const char *outputFilename = "../../DEMs/Output/parallel_flow_accum.tif"; //TODO should fix abs paths
    //Geotiff Driver
    GDALDriver *poDriver = GetGDALDriverManager()->GetDriverByName("GTiff");
    //32int Empty raster with same dims as input
    GDALDataset *flowAccumDataset = poDriver->Create(outputFilename,
                                                    D8Dataset->GetRasterXSize(),
                                                    D8Dataset->GetRasterYSize(),
                                                    1, GDT_Int32, NULL);

    //Raster size to use with Malloc and device mem
    int width = D8Dataset->GetRasterXSize();
    int height = D8Dataset->GetRasterYSize();
    bool *stopFlag = new bool;
    float *flowDir = (float *)CPLMalloc(sizeof(float) * width * height);
    

    //populate demData dynamically allocated memory
    CPLErr err = D8Dataset->GetRasterBand(1)->RasterIO(GF_Read, 0, 0, width, height, flowDir, width, height, GDT_Float32, 0, 0);
    if (err != CE_None){
        std::cerr << "Error reading DEM data: " << CPLGetLastErrorMsg() << std::endl;
        return -1;
    }

    //allocating device mem
    int *d_oldFlow, *d_newFlow, *d_flowDir;
    bool *d_stopFlag;

    
    hipMalloc(&d_oldFlow, width * height * sizeof(int));
    hipMalloc(&d_newFlow, width * height * sizeof(int));
    hipMalloc(&d_flowDir, width * height * sizeof(int));
    hipMalloc(&d_stopFlag, sizeof(bool));
    
    //copy flow direction data to device
    hipError_t memcpy_err_flowDir = hipMemcpy(d_flowDir, flowDir, sizeof(int) * width * height, hipMemcpyHostToDevice);
    if (memcpy_err_flowDir != hipSuccess){
        std::cerr << "Error copying data to device: " << hipGetErrorString(memcpy_err_flowDir) << std::endl;
        return -1;
    }
    int* hostOldFlow = new int[width * height];
    for (int i = 0; i < width * height; ++i) {
        hostOldFlow[i] = 1;
    }

    hipMemcpy(d_oldFlow, hostOldFlow, width * height * sizeof(int), hipMemcpyHostToDevice);
    hipMemset(d_newFlow, 0, sizeof(int) * width *height);
    //define grid and block size
    dim3 blockSize(BLOCK_SIZE, BLOCK_SIZE);
    dim3 gridSize((width + blockSize.x - 1) / blockSize.x, (height + blockSize.y - 1) / blockSize.y);


    //call kernel 
    do{
        *stopFlag = false;
        hipMemcpy(d_stopFlag, stopFlag, sizeof(bool), hipMemcpyHostToDevice);
        flowAccumKernel<<<gridSize, blockSize>>>(d_oldFlow, d_newFlow, d_flowDir, d_stopFlag, height, width);
        hipDeviceSynchronize();
        hipMemcpy(stopFlag, d_stopFlag, sizeof(bool), hipMemcpyDeviceToHost);
        int *temp = d_oldFlow;
        d_oldFlow = d_newFlow;
        d_newFlow = temp;

        hipMemset(d_newFlow, 0, sizeof(int) * width * height);
    } while (*stopFlag);
    int *hostflowAccumulationData = (int *)CPLMalloc(sizeof(int) * width * height);
    hipDeviceSynchronize();
    hipMemcpy(hostflowAccumulationData, d_oldFlow, sizeof(int) * width * height, hipMemcpyDeviceToHost);
    err = flowAccumDataset->GetRasterBand(1)->RasterIO(GF_Write, 0, 0, width, height,
        hostflowAccumulationData, width, height, GDT_Int32, 0, 0);
    if (err != CE_None) {
        std::cerr << "Error writing flow accumulation data: " << CPLGetLastErrorMsg() << std::endl;
        return -1;
    }
    
    
    hipFree(d_oldFlow);
    hipFree(d_newFlow);
    hipFree(d_flowDir);
    hipFree(d_stopFlag);
    CPLFree(flowDir);
    GDALClose(D8Dataset);
    GDALClose(flowAccumDataset);
    return 0;


}