#include "hip/hip_runtime.h"
#include <iostream>
#include <gdal_priv.h>
#include <hip/hip_runtime.h>
#include <cmath>
#include <cstdio>
#include <cstdlib>
#include <queue>
#include <vector>
#include <limits.h>
#include <float.h>

#define BLOCK_DIM_X 8
#define BLOCK_DIM_Y 8
#define HEIGHT_CONST 0.1f

// ./pit_filling ../../DEMs/092F.tif ../../DEMs/Output/092F_filled.tif
__global__ void identifyAndFillPits(float* dem, int* numPits, int width, int height, float hc){
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;


    if (x < 1 || x >= width - 1 || y < 1 || y >= height - 1) return; // skip boundary 

     __shared__ float sharedDem[BLOCK_DIM_Y + 2][BLOCK_DIM_X + 2];

    int tx = threadIdx.x + 1;
    int ty = threadIdx.y + 1;

    
    if (x < width && y < height){
        sharedDem[ty][tx] = dem[y * width + x];

        //left padding 
        if (threadIdx.x == 0 && x > 0){
            sharedDem[ty][0] = dem[y * width + (x - 1)];
        }
        //right padding
        if (threadIdx.x == blockDim.x - 1 && x < width - 1) {
            sharedDem[ty][tx + 1] = dem[y * width + (x + 1)];
        }
        //top padding
        if (threadIdx.y == 0 && y > 0){ 
            sharedDem[0][tx] = dem[(y - 1) * width + x];
        }
        //bottom padding
        if (threadIdx.y == blockDim.y - 1 && y < height - 1){
            sharedDem[ty + 1][tx] = dem[(y + 1) * width + x];
        }
    }
    __syncthreads();

    float curElev = static_cast<float>(sharedDem[ty][tx]);
    bool isPit = true;
    float lowestNeighbour = FLT_MAX;

    const int offsetX[8] = { -1, 0, 1, 0, -1, 1, 1, -1 };
    const int offsetY[8] = { 0, -1, 0, 1, -1, -1, 1, 1 };

    for (int i = 0; i < 8; i++){
        int neighbourElev = sharedDem[ty + offsetY[i]][tx + offsetX[i]];

        if (neighbourElev <= curElev){
            isPit = false;
            break;
        }else{
            if (neighbourElev < lowestNeighbour){
                lowestNeighbour = neighbourElev;
            }
        }
    }
    // if pit, store elev
    if (isPit){
        int count = atomicAdd(numPits, 1);
        dem[y * width + x] = static_cast<float>(lowestNeighbour) + hc;
    }
}
int main(int argc, char* argv[]){
    if (argc < 3){
        std::cout << "Please provide a filepath for input and output raster" << std::endl;
        return -1;
    }

    // register drivers to open raster data
    GDALAllRegister();
    
    // Open DEM dataset
    const char* input = argv[1];
    GDALDataset* demDataset  = (GDALDataset*) GDALOpen(input, GA_ReadOnly);

    if (demDataset == nullptr){
        std::cerr << "Error opening DEM file" << std::endl;
        return -1;
    }

    const char* projection = demDataset->GetProjectionRef();
    double geoTransform[6];

    if (demDataset->GetGeoTransform(geoTransform) != CE_None){
        std::cerr << "Error reading geo-transform" << std::endl;
        GDALClose(demDataset);
        return -1;
    }

    //create output raster for pit filling
    const char *outputFilename = argv[2];
    //Geotiff Driver
    GDALDriver *poDriver = GetGDALDriverManager()->GetDriverByName("GTiff");
    //32int Empty raster with same dims as input
    GDALDataset *pitFillingDataset = poDriver->Create(outputFilename,
                                                    demDataset->GetRasterXSize(),
                                                    demDataset->GetRasterYSize(),
                                                    1, GDT_Float32, NULL);

    pitFillingDataset->SetProjection(projection);
    pitFillingDataset->SetGeoTransform(geoTransform);

    //Raster size to use with Malloc and device mem
    int width = demDataset->GetRasterXSize();
    int height = demDataset->GetRasterYSize();
    float *demData = (float *)CPLMalloc(sizeof(float) * width * height);

    //populate demData dynamically allocated memory
    CPLErr err = demDataset->GetRasterBand(1)->RasterIO(GF_Read, 0, 0, width, height, demData, width, height, GDT_Float32, 0, 0);
    if (err != CE_None){
        std::cerr << "Error reading DEM data: " << CPLGetLastErrorMsg() << std::endl;
        GDALClose(demDataset);
        return -1;
    }

    //allocating device mem
    float *d_dem;

    if (hipMalloc(&d_dem, sizeof(float) * width * height) != hipSuccess) {
        std::cerr << "Error allocating memory for DEM on device." << std::endl;
        return -1;
    }

    //copy DEM data to device
    hipError_t memcpy_err = hipMemcpy(d_dem, demData, sizeof(float) * width * height, hipMemcpyHostToDevice);
    if (memcpy_err != hipSuccess){
        std::cerr << "Error copying data to device: " << hipGetErrorString(memcpy_err) << std::endl;
        return -1;
    }

    int* d_numPits;
    int numPits = 0;
    hipMalloc(&d_numPits, sizeof(int));
    hipMemset(d_numPits, 0, sizeof(int));

    //define grid and block size
    dim3 blockSize(BLOCK_DIM_X,BLOCK_DIM_Y);
    dim3 gridSize((width + BLOCK_DIM_X - 1) / BLOCK_DIM_X, (height + BLOCK_DIM_Y - 1) / BLOCK_DIM_Y);

    identifyAndFillPits<<<gridSize, blockSize>>>(d_dem, d_numPits, width, height, HEIGHT_CONST);
    hipError_t kernel_err = hipGetLastError();
    if (kernel_err != hipSuccess){
        std::cerr << "Kernel Launch failed: " << hipGetErrorString(kernel_err) << std::endl;
        return -1;
    }
    hipDeviceSynchronize();

    // copy pit count from device to host
    hipMemcpy(&numPits, d_numPits, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(demData, d_dem, sizeof(float) * width * height, hipMemcpyDeviceToHost);

    // Write pit filling data to the output dataset
    err = pitFillingDataset->GetRasterBand(1)->RasterIO(GF_Write, 0, 0, width, height, demData, width, height, GDT_Float32, 0, 0);
    if (err != CE_None) {
        std::cerr << "Error writing pit filling data: " << CPLGetLastErrorMsg() << std::endl;
        return -1;
    }

    hipFree(d_dem);
    hipFree(d_numPits);
    CPLFree(demData);
    GDALClose(demDataset);
    GDALClose(pitFillingDataset);

    std::cout << "Number of pits filled: " << numPits << std::endl;
    std::cout << "Pit filling completed and output written to " << outputFilename << std::endl;
    return 0;
}