#include "hip/hip_runtime.h"
#include <iostream>
#include <gdal_priv.h>
#include <hip/hip_runtime.h>
#include <cmath>
#include <cstdio>
#include <cstdlib>
#include <float.h>

#define FLOW_NODATA -1
#define HEIGHT_CONST 0.1f
#define BLOCK_DIM_X 8
#define BLOCK_DIM_Y 8

__constant__ int offsetX[8] = { -1, 0, 1, 0, -1, 1, 1, -1 };
__constant__ int offsetY[8] = { 0, -1, 0, 1, -1, -1, 1, 1 };
__constant__ int direction[8] = { 8, 2, 4, 6, 1, 3, 5, 7 }; 

__global__ void pitFillFlowDirectionKernel(const float* dem, int* flow_dir, int* numPits, int width, int height, float hc) {
    //unique thread index
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x <= 0 || x >= width - 1 || y <= 0 || y >= height - 1) return; //skip boundary 
    
    __shared__ float sharedDem[BLOCK_DIM_X + 2][BLOCK_DIM_Y + 2]; //extra column padding

    //local indices for use with padded shared
    int tx = threadIdx.x + 1;
    int ty = threadIdx.y + 1;

    //copy center pixel to shared
    sharedDem[ty][tx] = dem[y * width + x];

    //left padding 
    if (threadIdx.x == 0 && x > 0){
        sharedDem[ty][0] = dem[y * width + (x - 1)];
    }
    //right padding
    if (threadIdx.x == blockDim.x - 1 && x < width - 1) {
        sharedDem[ty][tx + 1] = dem[y * width + (x + 1)];
    }
    //top padding
    if (threadIdx.y == 0 && y > 0){ 
        sharedDem[0][tx] = dem[(y - 1) * width + x];
    }
    //bottom padding
    if (threadIdx.y == blockDim.y - 1 && y < height - 1){
        sharedDem[ty + 1][tx] = dem[(y + 1) * width + x];
    }
    //top left corner
    if (threadIdx.x == 0 && threadIdx.y == 0 && x > 0 && y > 0) {
    sharedDem[0][0] = dem[(y - 1) * width + (x - 1)];
    }
    //top right corner
    if (threadIdx.x == blockDim.x - 1 && threadIdx.y == 0 && x < width - 1 && y > 0) {
        sharedDem[0][tx + 1] = dem[(y - 1) * width + (x + 1)];
    }
    //bottom left corner
    if (threadIdx.x == 0 && threadIdx.y == blockDim.y - 1 && x > 0 && y < height - 1) {
        sharedDem[ty + 1][0] = dem[(y + 1) * width + (x - 1)];
    }
    // bottom right corner
    if (threadIdx.x == blockDim.x - 1 && threadIdx.y == blockDim.y - 1 && x < width - 1 && y < height - 1) {
        sharedDem[ty + 1][tx + 1] = dem[(y + 1) * width + (x + 1)];
    }
    __syncthreads();

    float centre = sharedDem[ty][tx]; //get dem value at current pixel
    float lowest = centre;
    int dir = FLOW_NODATA;
    float lowestNeighbour = FLT_MAX;
    bool isPit = true;

    //pit check
    for (int i = 0; i < 8; i++) {
        float neighbourElev = sharedDem[ty + offsetY[i]][tx + offsetX[i]];
        if (neighbourElev <= centre){
            isPit = false;
            break;
        }else{
            if (neighbourElev < lowestNeighbour){
                lowestNeighbour = neighbourElev;
            }
        }
    }

    //keep track of pitcount in shared to reduce writes to global
    __shared__ int localPits;
    if (threadIdx.x == 0 && threadIdx.y == 0) localPits = 0;
    __syncthreads();

    if (isPit){
        atomicAdd(&localPits, 1);
        //update sharedDem with pit filling step for flow direction calculation
        sharedDem[ty][tx] = lowestNeighbour + hc;
    }
    __syncthreads();

    if (threadIdx.x == 0 && threadIdx.y == 0){
        atomicAdd(numPits, localPits);
    }

    for (int i = 0; i < 8; i++) {
        float neighbour = sharedDem[ty + offsetY[i]][tx + offsetX[i]];

        if (neighbour < lowest){
            lowest = neighbour;
            dir = direction[i];
        }
    }
    if (dir != FLOW_NODATA){
        flow_dir[y * width + x] = dir;
    }
}

void cleanup(float* demData, int* flowDirData, float* d_demData, int* d_flowDirData, int* d_numPits) {
    //helped function for cleaup of dynamically allocated memory
    if (demData) CPLFree(demData);
    if (flowDirData) CPLFree(flowDirData);
    if (d_demData) hipFree(d_demData);
    if (d_flowDirData) hipFree(d_flowDirData);
    if (d_numPits) hipFree(d_numPits);
}

int main(int argc, char* argv[]) {
    //checks for input file passed as arg
    if (argc < 3){
        std::cout << "Please provide a filepath for input and output raster" << std::endl;
        return -1;
    }

    // register drivers to open raster data
    GDALAllRegister();
    
    // Open DEM dataset
    const char* input = argv[1];
    GDALDataset* demDataset  = (GDALDataset*) GDALOpen(input, GA_ReadOnly);

    if (demDataset == nullptr) {
        std::cerr << "Error opening DEM file." << std::endl;
        return -1;
    }

    //get projection from input raster
    const char* projection = demDataset->GetProjectionRef();
    if (projection == nullptr){
        std::cerr << "Error: Could not retrieve projection from DEM Dataset." << std::endl;
        GDALClose(demDataset);
        return -1;
    }
    double geoTransform[6];

    if (demDataset->GetGeoTransform(geoTransform) != CE_None){
        std::cerr << "Error reading geo-transform" << std::endl;
        GDALClose(demDataset);
        return -1;
    }

    //create output raster for flow direction
    const char *outputFilename = argv[2];
    //Geotiff Driver
    GDALDriver *poDriver = GetGDALDriverManager()->GetDriverByName("GTiff");
    //32int Empty raster with same dims as input
    GDALDataset *flowDirDataset = poDriver->Create(outputFilename,
                                                    demDataset->GetRasterXSize(),
                                                    demDataset->GetRasterYSize(),
                                                    1, GDT_Int32, NULL);

    flowDirDataset->SetProjection(projection);
    flowDirDataset->SetGeoTransform(geoTransform);

    //Raster size to use with Malloc and device mem
    int width = demDataset->GetRasterXSize();
    int height = demDataset->GetRasterYSize();
    float *demData = (float *)CPLMalloc(sizeof(float) * width * height);

    //populate demData dynamically allocated memory
    CPLErr err = demDataset->GetRasterBand(1)->RasterIO(GF_Read, 0, 0, width, height, demData, width, height, GDT_Float32, 0, 0);
    if (err != CE_None){
        std::cerr << "Error reading DEM data: " << CPLGetLastErrorMsg() << std::endl;
        cleanup(demData, nullptr, nullptr, nullptr, nullptr);
        return -1;
    }

    int *flowDirData = (int *)CPLMalloc(sizeof(int) * width * height);

    //create cuda stream
    hipStream_t stream;
    hipError_t stream_err =hipStreamCreate(&stream);
    if (stream_err != hipSuccess){
        std::cerr << "Error creating CUDA stream: " <<hipGetErrorString(stream_err) << std::endl;
        cleanup(demData, flowDirData, nullptr, nullptr, nullptr);
        return -1;
    }

    int numPits = 0;
    int* d_numPits; 
    if (hipMalloc(&d_numPits, sizeof(int)) != hipSuccess){
        std::cerr << "Error allocationg memory for Pit Count on device." << std::endl;
        cleanup(demData, flowDirData, nullptr, nullptr, d_numPits);
        return -1;
    }
    hipMemset(d_numPits, 0, sizeof(int));

    //allocating device mem
    float *d_demData;
    if (hipMalloc(&d_demData, sizeof(float) * width * height) != hipSuccess) {
        std::cerr << "Error allocating memory for DEM on device." << std::endl;
        cleanup(demData, flowDirData, d_demData, nullptr, d_numPits);
        return -1;
    }

    int *d_flowDirData;
    if (hipMalloc(&d_flowDirData, sizeof(int) * width * height) != hipSuccess) {
        std::cerr << "Error allocating memory for flow direction on device." << std::endl;
        cleanup(demData, flowDirData, d_demData, d_flowDirData, d_numPits);
        return -1;
    }

    //copy DEM data to device
    hipError_t memcpy_err = hipMemcpyAsync(d_demData, demData, sizeof(int) * width * height, hipMemcpyHostToDevice);
    if (memcpy_err != hipSuccess){
        std::cerr << "Error copying data to device: " << hipGetErrorString(memcpy_err) << std::endl;
        return -1;
    }

    //define grid and block size
    dim3 blockSize(BLOCK_DIM_X, BLOCK_DIM_Y);
    dim3 gridSize((width + blockSize.x - 1) / blockSize.x, (height + blockSize.y - 1) / blockSize.y); //dynamic grid allocation based on input size

    // Launch the CUDA kernel
    pitFillFlowDirectionKernel<<<gridSize, blockSize, 0, stream>>>(d_demData, d_flowDirData, d_numPits, width, height, HEIGHT_CONST);

    hipError_t kernel_err = hipGetLastError();
    if (kernel_err != hipSuccess){
        std::cerr << "Cuda kernel launch error: " << hipGetErrorString(kernel_err) << std::endl;
        return -1;
    }

    // Copy flow direction data back to host
    memcpy_err = hipMemcpyAsync(flowDirData, d_flowDirData, sizeof(int) * width * height, hipMemcpyDeviceToHost);
    if (memcpy_err != hipSuccess){
        std::cerr << "Error copying data to device: " << hipGetErrorString(memcpy_err) << std::endl;
        return -1;
    }

    //copy numpits back to device
    memcpy_err = hipMemcpyAsync(&numPits, d_numPits, sizeof(int), hipMemcpyDeviceToHost, stream);
    if (memcpy_err != hipSuccess){
        std::cerr << "Error copying data to device: " << hipGetErrorString(memcpy_err) << std::endl;
        return -1;
    }

    hipStreamSynchronize(stream);
    
    // Write flow direction data to the output dataset
    err = flowDirDataset->GetRasterBand(1)->RasterIO(GF_Write, 0, 0, width, height, flowDirData, width, height, GDT_Int32, 0, 0);
    if (err != CE_None) {
        std::cerr << "Error writing flow direction data: " << CPLGetLastErrorMsg() << std::endl;
        return -1;
    }

    // Cleanup
    cleanup(demData, flowDirData, d_demData, d_flowDirData, d_numPits);
    GDALClose(demDataset);
    GDALClose(flowDirDataset);

    std::cout << "Flow direction calculated and saved to " << outputFilename << std::endl;
    return 0;
}
